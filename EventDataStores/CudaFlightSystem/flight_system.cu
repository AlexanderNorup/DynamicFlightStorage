#include "hip/hip_runtime.h"
#include "flight_system.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/remove.h>
#include <iostream>

// CUDA kernel to update specific flights
__global__ void updateFlightsKernel(Flight* flights, int* indices, Vec3* newPositions, int updateCount) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < updateCount) {
		int flightIdx = indices[idx];
		flights[flightIdx].position = newPositions[idx];
	}
}

// CUDA kernel to check collisions between flights and a bounding box
__global__ void checkCollisionsKernel(Flight* flights, int numFlights,
	int* indices, BoundingBox box, int* collisionResults) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < numFlights) {
		int flightIdx = indices[idx];
		Vec3 pos = flights[flightIdx].position;

		// Check if point is inside the bounding box
		bool collision =
			(pos.x >= box.min.x) && (pos.x <= box.max.x) &&
			(pos.y >= box.min.y) && (pos.y <= box.max.y) &&
			(pos.z >= box.min.z) && (pos.z <= box.max.z);

		collisionResults[flightIdx] = collision ? 1 : 0;
	}
}

// Custom comparison functor for sorting by x position
struct CompareByX {
	Flight* flights;

	CompareByX(Flight* _flights) : flights(_flights) {}

	__host__ __device__ bool operator()(int a, int b) const {
		return flights[a].position.x < flights[b].position.x;
	}
};

// Constructor - initialize member variables
FlightSystem::FlightSystem()
	: d_flights(nullptr), d_indices(nullptr), d_collisionResults(nullptr),
	numFlights(0), allocatedFlights(0), initialized(false), deviceId(0) {
	// Get the current CUDA device
	hipGetDevice(&deviceId);
}

// Destructor - cleanup CUDA resources
FlightSystem::~FlightSystem() {
	cleanup();
}

// Allocate or reallocate device memory
bool FlightSystem::allocateDeviceMemory(int requiredSize) {
	// If we already have enough space, no need to reallocate
	if (allocatedFlights >= requiredSize && d_flights != nullptr &&
		d_indices != nullptr && d_collisionResults != nullptr) {
		return true;
	}

	// Calculate new allocation size (with some extra space for future additions)
	int newSize = requiredSize * 1.5; // Allocate 50% extra space
	if (newSize < 100) newSize = 100; // Minimum allocation

	// Allocate new memory
	Flight* new_d_flights = nullptr;
	int* new_d_indices = nullptr;
	int* new_d_collisionResults = nullptr;

	hipError_t error = hipMalloc(&new_d_flights, newSize * sizeof(Flight));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for flights: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	error = hipMalloc(&new_d_indices, newSize * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for indices: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(new_d_flights);
		return false;
	}

	error = hipMalloc(&new_d_collisionResults, newSize * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for collision results: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(new_d_flights);
		hipFree(new_d_indices);
		return false;
	}

	// If we're reallocating, copy existing data to new memory
	if (d_flights != nullptr && numFlights > 0) {
		hipMemcpy(new_d_flights, d_flights, numFlights * sizeof(Flight), hipMemcpyDeviceToDevice);
		hipMemcpy(new_d_indices, d_indices, numFlights * sizeof(int), hipMemcpyDeviceToDevice);

		// Free old memory
		hipFree(d_flights);
		hipFree(d_indices);
		hipFree(d_collisionResults);
	}

	// Set the new pointers
	d_flights = new_d_flights;
	d_indices = new_d_indices;
	d_collisionResults = new_d_collisionResults;
	allocatedFlights = newSize;

	return true;
}

// Initialize with flights from host
bool FlightSystem::initialize(Flight* hostFlights, int count) {
	// Clean up previous allocation if any
	cleanup();

	if (count <= 0 || hostFlights == nullptr) {
		std::cerr << "Invalid flight data provided for initialization" << std::endl;
		return false;
	}

	numFlights = count;

	// Allocate device memory
	if (!allocateDeviceMemory(count)) {
		return false;
	}

	// Copy flights to device
	hipError_t error = hipMemcpy(d_flights, hostFlights, numFlights * sizeof(Flight), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy flights to device: "
			<< hipGetErrorString(error) << std::endl;
		cleanup();
		return false;
	}

	// Initialize indices and sort flights
	sortFlightsByX();

	initialized = true;
	return true;
}

// Add new flights to the system
bool FlightSystem::addFlights(Flight* newFlights, int count) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (count <= 0 || newFlights == nullptr) {
		std::cerr << "Invalid flight data provided for addition" << std::endl;
		return false;
	}

	// Check if we need to reallocate memory
	int newTotal = numFlights + count;
	if (newTotal > allocatedFlights) {
		if (!allocateDeviceMemory(newTotal)) {
			return false;
		}
	}

	// Copy new flights to the end of existing flights
	hipError_t error = hipMemcpy(d_flights + numFlights, newFlights, count * sizeof(Flight), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy new flights to device: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	// Update flight count
	numFlights = newTotal;

	// Re-sort flights by X coordinate
	sortFlightsByX();

	return true;
}

// Remove flights by indices
bool FlightSystem::removeFlights(int* indices, int count) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (count <= 0 || indices == nullptr) {
		std::cerr << "Invalid indices provided for removal" << std::endl;
		return false;
	}

	// Create a temporary host array of all flights
	Flight* hostFlights = new Flight[numFlights];
	if (!hostFlights) {
		std::cerr << "Failed to allocate host memory for flight removal" << std::endl;
		return false;
	}

	// Copy flights from device to host
	hipError_t error = hipMemcpy(hostFlights, d_flights, numFlights * sizeof(Flight), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy flights to host for removal: " << hipGetErrorString(error) << std::endl;
		delete[] hostFlights;
		return false;
	}

	// Create a flagged array to mark flights for removal
	bool* toRemove = new bool[numFlights]();
	for (int i = 0; i < count; i++) {
		if (indices[i] >= 0 && indices[i] < numFlights) {
			toRemove[indices[i]] = true;
		}
		else {
			std::cerr << "Invalid flight index for removal: " << indices[i] << std::endl;
		}
	}

	// Create a new array without removed flights
	int newCount = 0;
	Flight* newFlights = new Flight[numFlights];

	for (int i = 0; i < numFlights; i++) {
		if (!toRemove[i]) {
			newFlights[newCount++] = hostFlights[i];
		}
	}

	// Clean up temporary arrays
	delete[] hostFlights;
	delete[] toRemove;

	// Reinitialize with the new array
	bool result = initialize(newFlights, newCount);

	// Clean up the new array
	delete[] newFlights;

	return result;
}

// Update specific flights with new positions
bool FlightSystem::updateFlights(int* indices, Vec3* newPositions, int updateCount) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (updateCount <= 0 || indices == nullptr || newPositions == nullptr) {
		std::cerr << "Invalid data provided for flight update" << std::endl;
		return false;
	}

	// Allocate device memory for indices and new positions
	int* d_updateIndices;
	Vec3* d_newPositions;

	hipError_t error = hipMalloc(&d_updateIndices, updateCount * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for update indices: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	error = hipMalloc(&d_newPositions, updateCount * sizeof(Vec3));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for new positions: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		return false;
	}

	// Copy indices and new positions to device
	hipMemcpy(d_updateIndices, indices, updateCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_newPositions, newPositions, updateCount * sizeof(Vec3), hipMemcpyHostToDevice);

	// Launch kernel to update flights
	int blockSize = 256;
	int numBlocks = (updateCount + blockSize - 1) / blockSize;

	updateFlightsKernel << <numBlocks, blockSize >> > (
		d_flights, d_updateIndices, d_newPositions, updateCount);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Check for errors
	error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "Error updating flights: " << hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		hipFree(d_newPositions);
		return false;
	}

	// Free temporary device memory
	hipFree(d_updateIndices);
	hipFree(d_newPositions);

	// Re-sort flights by X coordinate after update
	sortFlightsByX();

	return true;
}

// Sort flights by X coordinate for efficient sweep
void FlightSystem::sortFlightsByX() {
	// Initialize indices
	thrust::sequence(thrust::device, d_indices, d_indices + numFlights, 0);

	// Sort flights by their x-coordinate
	thrust::sort(thrust::device, d_indices, d_indices + numFlights,
		CompareByX(d_flights));
}

// Detect collisions with a bounding box
bool FlightSystem::detectCollisions(const BoundingBox& box, int* collisionResults) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	// Launch collision detection kernel
	int blockSize = 256;
	int numBlocks = (numFlights + blockSize - 1) / blockSize;

	checkCollisionsKernel << <numBlocks, blockSize >> > (
		d_flights, numFlights, d_indices, box, d_collisionResults);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Check for errors
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "Error detecting collisions: " << hipGetErrorString(error) << std::endl;
		return false;
	}

	// Copy results back to host
	error = hipMemcpy(collisionResults, d_collisionResults, numFlights * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy collision results to host: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	return true;
}

// Free all allocated device memory
void FlightSystem::cleanup() {
	if (d_flights) {
		hipFree(d_flights);
		d_flights = nullptr;
	}

	if (d_indices) {
		hipFree(d_indices);
		d_indices = nullptr;
	}

	if (d_collisionResults) {
		hipFree(d_collisionResults);
		d_collisionResults = nullptr;
	}

	initialized = false;
	numFlights = 0;
	allocatedFlights = 0;
}
#include "hip/hip_runtime.h"
#include "flight_system.h"
#include "console_colors.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <thrust/extrema.h>
#include <iostream>

// CUDA kernel to update specific flights
__global__ void updateFlightsKernel(Flight* flights, int* indices, int* zData,
	FlightPosition* newPositions, int* newDurations, int* newZData, int* zDifferentFlag, int updateCount) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < updateCount) {
		int flightIdx = indices[idx];
		int oldZOffset = flights[flightIdx].position.zOffset;
		int oldZLength = flights[flightIdx].position.zLength;

		if (oldZLength != newPositions[idx].zLength && zDifferentFlag[0] == 0) {
			// We can't just update in place, since new z-length is different from what we have allocated for
			// We could somewhat easily fix it in the case where the new zLength is less than the allocated one.
			// But when the new zLength is bigger, we have no choice but to rebuild the system.
			zDifferentFlag[0] = 1;
		}

		int* zAddr = oldZOffset + zData;
		for (int i = 0; i < oldZLength; i++) {
			zAddr[i] = newZData[newPositions[idx].zOffset + i];
		}

		flights[flightIdx].position = newPositions[idx];
		flights[flightIdx].position.zLength = oldZLength;
		flights[flightIdx].position.zOffset = oldZOffset;
		flights[flightIdx].flightDuration = newDurations[idx];
		flights[flightIdx].isRecalculating = false;
	}
}

// CUDA kernel to check collisions between flights and a bounding box
__global__ void checkCollisionsKernel(Flight* flights, int numFlights,
	int* indices, int* zValues, BoundingBox box, int offset, bool setRecalculating, int* collisionResults) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx + offset < numFlights) {
		int flightIdx = indices[idx + offset];
		FlightPosition dep = flights[flightIdx].position;
		int duration = flights[flightIdx].flightDuration;

		collisionResults[idx + offset] = INT_MIN; // Default to no collision

		if (flights[flightIdx].isRecalculating) {
			return;
		}

		// We must check if the box intersects with the flight.
		// The flight is a LINE in 3D space made up of points (dep, dest) where dest.x = dep.x + duration
		// Y and Z coordinates are the same. So if they're not in the box, the flight doesn't intersect the box.

		bool yCollision = (dep.y >= box.min.y) && (dep.y <= box.max.y);

		if (!yCollision) {
			return;
		}

		bool zCollision = false;
		int* addr = dep.zOffset + zValues;
		for (int i = 0; i < dep.zLength; i++) {
			if (addr[i] >= box.min.z && addr[i] <= box.max.z) {
				zCollision = true;
				break;
			}
		}

		if (!zCollision) {
			return;
		}

		float xDest = dep.x + duration;
		bool collision =
			((dep.x >= box.min.x) && (dep.x <= box.max.x)) // Checks if flight starts inside the box
			|| (dep.x < box.min.x && xDest > box.min.x); // Checks if flight intersects the box

		if (collision) {
			if (setRecalculating) {
				flights[flightIdx].isRecalculating = true;
			}
			collisionResults[idx] = flights[flightIdx].id;
		}
	}
}

// Custom comparison functor for sorting by x position
struct CompareByX {
	Flight* flights;

	CompareByX(Flight* _flights) : flights(_flights) {}

	__host__ __device__ bool operator()(int a, int b) const {
		return flights[a].position.x < flights[b].position.x;
	}
};

// Custom compare function for lower/upper bound search to compare x-coordinate
struct CompareToLowerX {
	Flight* flights;

	CompareToLowerX(Flight* _flights) : flights(_flights) {}

	__host__ __device__ bool operator()(int idx, int val) const {
		return flights[idx].position.x < val;
	}
};

struct CompareByDuration {
	__host__ __device__ bool operator()(Flight a, Flight b)
	{
		return a.flightDuration < b.flightDuration;
	}
};


// Constructor - initialize member variables
FlightSystem::FlightSystem()
	: d_flights(nullptr), d_indices(nullptr), d_collisionResults(nullptr),
	numFlights(0), allocatedFlights(0), initialized(false), deviceId(0) {
	// Get the current CUDA device
	hipGetDevice(&deviceId);
}

// Destructor - cleanup CUDA resources
FlightSystem::~FlightSystem() {
	cleanup();
}

void FlightSystem::debug() {
	std::cout << "Flight count: " << numFlights << std::endl;
	std::cout << "Allocated flights: " << allocatedFlights << std::endl;

	std::vector<int> flightIndices(numFlights);
	std::vector<Flight> flights(numFlights);
	std::vector<int> zData(d_flightZData.size());
	hipMemcpy(flightIndices.data(), d_indices, numFlights * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(flights.data(), d_flights, numFlights * sizeof(Flight), hipMemcpyDeviceToHost);
	hipMemcpy(zData.data(), thrust::raw_pointer_cast(d_flightZData.data()), numFlights * sizeof(int), hipMemcpyDeviceToHost);

	flightIndices.clear();
	flights.clear();
	zData.clear();
}

// Allocate or reallocate device memory
bool FlightSystem::allocateDeviceMemory(int requiredSize) {
	// If we already have enough space, no need to reallocate
	if (allocatedFlights >= requiredSize && d_flights != nullptr &&
		d_indices != nullptr && d_collisionResults != nullptr) {
		return true;
	}

	// Calculate new allocation size (with some extra space for future additions)
	int newSize = requiredSize * 1.5; // Allocate 50% extra space
	if (newSize < 100) newSize = 100; // Minimum allocation

	// Allocate new memory
	Flight* new_d_flights = nullptr;
	int* new_d_indices = nullptr;
	int* new_d_collisionResults = nullptr;

	hipError_t error = hipMalloc(&new_d_flights, newSize * sizeof(Flight));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for flights: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	error = hipMalloc(&new_d_indices, newSize * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for indices: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(new_d_flights);
		return false;
	}

	error = hipMalloc(&new_d_collisionResults, newSize * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for collision results: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(new_d_flights);
		hipFree(new_d_indices);
		return false;
	}

	// If we're reallocating, copy existing data to new memory
	if (d_flights != nullptr && numFlights > 0) {
		hipMemcpy(new_d_flights, d_flights, numFlights * sizeof(Flight), hipMemcpyDeviceToDevice);
		hipMemcpy(new_d_indices, d_indices, numFlights * sizeof(int), hipMemcpyDeviceToDevice);
	}

	// Free old memory
	hipFree(d_flights);
	hipFree(d_indices);
	hipFree(d_collisionResults);

	// Set the new pointers
	d_flights = new_d_flights;
	d_indices = new_d_indices;
	d_collisionResults = new_d_collisionResults;
	allocatedFlights = newSize;

	return true;
}

// Makes a Flight* ready to be copied into device memory. 
void FlightSystem::copyZDataToDeviceManaged(Flight* flights, int count) {
	// Copy all the airports to device
	std::vector<int> airports;
	airports.reserve(count * 2.5);
	for (int i = 0; i < count; i++) {
		for (int j = 0; j < flights[i].position.zLength; j++) {
			airports.push_back(flights[i].position.z[j]);
		}
	}

	if (airports.size() == 0) {
		return;
	}

	int newSize = d_flightZData.size() + airports.size();
	if (newSize < d_flightZData.capacity()) {
		d_flightZData.reserve(newSize * 1.5);
	}
	int previousSize = d_flightZData.size();
	d_flightZData.resize(newSize);

	int* newDataStart = thrust::raw_pointer_cast(d_flightZData.data()) + previousSize;
	hipMemcpy(newDataStart, airports.data(), airports.size() * sizeof(int), hipMemcpyHostToDevice);

	airports.clear();
	int counter = 0;
	for (int i = 0; i < count; i++) {
		if (flights[i].position.zLength <= 0) {
			continue;
		}
		flights[i].position.zOffset = previousSize + counter;
		counter += flights[i].position.zLength;
	}
}

// Initialize with flights from host
bool FlightSystem::initialize(Flight* hostFlights, int count) {
	// Clean up previous allocation if any
	cleanup();

	if (count < 0) {
		std::cerr << "Invalid flight count provided for initialization" << std::endl;
		return false;
	}

	numFlights = count;

	// Allocate device memory
	if (!allocateDeviceMemory(count)) {
		return false;
	}

	if (hostFlights != nullptr) {
		copyZDataToDeviceManaged(hostFlights, count);

		// Copy flights to device
		hipError_t error = hipMemcpy(d_flights, hostFlights, numFlights * sizeof(Flight), hipMemcpyHostToDevice);
		if (error != hipSuccess) {
			std::cerr << "Failed to copy flights to device: "
				<< hipGetErrorString(error) << std::endl;
			cleanup();
			return false;
		}

		// Initialize indices and sort flights
		indicesDirty = true;

		// Update the ID to index mapping
		updateIdToIndexMap();
	}

	initialized = true;
	return true;
}

// Add new flights to the system
bool FlightSystem::addFlights(Flight* newFlights, int count) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (count < 0 || newFlights == nullptr) {
		std::cerr << "Invalid flight data provided for addition" << std::endl;
		return false;
	}

	// Check if we need to reallocate memory
	int newTotal = numFlights + count;
	if (newTotal > allocatedFlights) {
		if (!allocateDeviceMemory(newTotal)) {
			return false;
		}
	}

	// Copy all the airports to device
	copyZDataToDeviceManaged(newFlights, count);

	// Copy new flights to the end of existing flights
	hipError_t error = hipMemcpy(d_flights + numFlights, newFlights, count * sizeof(Flight), hipMemcpyHostToDevice);

	if (error != hipSuccess) {
		std::cerr << "Failed to copy new flights to device: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	// Update flight count
	numFlights = newTotal;

	// Re-sort flights by X coordinate
	indicesDirty = true;

	flightIdToIndex.reserve(numFlights);
	for (int i = 0; i < count; i++) {
		flightIdToIndex[newFlights[i].id] = numFlights - count + i;
	}

	return true;
}

// Remove flights by ids
bool FlightSystem::removeFlights(int* ids, int count) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (count <= 0 || ids == nullptr) {
		std::cerr << "Invalid ids provided for removal" << std::endl;
		return false;
	}

	int* indices = new int[count];
	getIndicesFromIds(ids, count, indices);

	// Create a temporary host array of all flights
	Flight* hostFlights = new Flight[numFlights];
	if (!hostFlights) {
		std::cerr << "Failed to allocate host memory for flight removal" << std::endl;
		return false;
	}

	// Copy flights from device to host
	hipError_t error = hipMemcpy(hostFlights, d_flights, numFlights * sizeof(Flight), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy flights to host for removal: " << hipGetErrorString(error) << std::endl;
		delete[] hostFlights;
		return false;
	}

	// Create a flagged array to mark flights for removal
	bool* toRemove = new bool[numFlights]();
	for (int i = 0; i < count; i++) {
		if (indices[i] >= 0 && indices[i] < numFlights) {
			toRemove[indices[i]] = true;
		}
		else {
			std::cerr << "Invalid flight index for removal: " << indices[i] << std::endl;
		}
	}

	// Create a new array without removed flights
	int newCount = 0;
	Flight* newFlights = new Flight[numFlights];

	int* newAirports = new int[d_flightZData.size()];
	hipMemcpy(newAirports, thrust::raw_pointer_cast(d_flightZData.data()), d_flightZData.size() * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < numFlights; i++) {
		if (!toRemove[i]) {
			newFlights[newCount++] = hostFlights[i];
			if (newFlights[newCount - 1].position.zLength > 0) {
				newFlights[newCount - 1].position.z = newAirports + newFlights[newCount - 1].position.zOffset;
			}
			else {
				newFlights[newCount - 1].position.z = nullptr;
			}
		}
	}

	flightIdMapDirty = true;

	// Clean up temporary arrays
	delete[] hostFlights;
	delete[] toRemove;

	// Reinitialize with the new array
	bool result = initialize(newFlights, newCount);

	// Clean up the new array
	delete[] newFlights;
	delete[] newAirports;

	return result;
}

// Update specific flights with new positions
bool FlightSystem::updateFlights(int* ids, FlightPosition* newPositions, int* newDurations, int updateCount) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (updateCount <= 0 || ids == nullptr || newPositions == nullptr || newDurations == nullptr) {
		std::cerr << "Invalid data provided for flight update" << std::endl;
		return false;
	}

	// Allocate device memory for indices, new positions and durations
	int* d_updateIndices;
	FlightPosition* d_newPositions;
	int* d_newDurations;

	hipError_t error = hipMalloc(&d_updateIndices, updateCount * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for update indices: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	error = hipMalloc(&d_newPositions, updateCount * sizeof(FlightPosition));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for new positions: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		return false;
	}

	error = hipMalloc(&d_newDurations, updateCount * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for new durations: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		hipFree(d_newPositions);
		return false;
	}

	// Copy the airport data over

	std::vector<int> airports;
	int counter = 0;
	airports.reserve(updateCount * 2.5);
	for (int i = 0; i < updateCount; i++) {
		newPositions[i].zOffset = counter;
		counter += newPositions[i].zLength;
		for (int j = 0; j < newPositions[i].zLength; j++) {
			airports.push_back(newPositions[i].z[j]);
		}
	}

	int* d_newZData;
	error = hipMalloc(&d_newZData, airports.size() * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for new durations: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		hipFree(d_newPositions);
		hipFree(d_newDurations);
		return false;
	}

	int* d_zDifferentFlag;
	error = hipMalloc(&d_zDifferentFlag, sizeof(int));

	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for zDifferentFlag: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		hipFree(d_newPositions);
		hipFree(d_newDurations);
		hipFree(d_newZData);
		return false;
	}

	// Find the indicies
	std::vector<int> indices(updateCount);
	getIndicesFromIds(ids, updateCount, indices.data());

	// Copy indices and new positions to device
	hipMemcpy(d_updateIndices, indices.data(), updateCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_newPositions, newPositions, updateCount * sizeof(FlightPosition), hipMemcpyHostToDevice);
	hipMemcpy(d_newDurations, newDurations, updateCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_newZData, airports.data(), airports.size() * sizeof(int), hipMemcpyHostToDevice);
	thrust::fill(thrust::device, d_zDifferentFlag, d_zDifferentFlag + 1, 0); // Set to 0

	// Launch kernel to update flights
	int blockSize = 256;
	int numBlocks = (updateCount + blockSize - 1) / blockSize;

	int* oldZData = thrust::raw_pointer_cast(d_flightZData.data());

	updateFlightsKernel << <numBlocks, blockSize >> > (
		d_flights, d_updateIndices, oldZData, d_newPositions, d_newDurations, d_newZData, d_zDifferentFlag, updateCount);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Free temporary device memory
	hipFree(d_updateIndices);
	hipFree(d_newPositions);
	hipFree(d_newDurations);
	hipFree(d_newZData);

	// Check for errors
	error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "Error updating flights: " << hipGetErrorString(error) << std::endl;
		return false;
	}

	int zDifferentFlag;
	hipMemcpy(&zDifferentFlag, d_zDifferentFlag, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_zDifferentFlag);

	if (zDifferentFlag == 1) {
		std::cout << COLOR_YELLOW << "Warning: Slow flight update due to rebuilding of airport array" << COLOR_RESET << std::endl;
		// We got new flights with different amount of z-values which we don't have space for. 
		// We need to reinitialize the system. We copy all flights out of the GPU.
		// Manually update the flights, and then reinsert them.

		// Create a temporary host array of all flights
		Flight* hostFlights = new Flight[numFlights];
		if (!hostFlights) {
			std::cerr << "Failed to allocate host memory for flight updating" << std::endl;
			return false;
		}

		int* newAirports = new int[d_flightZData.size()];
		hipError_t errorA = hipMemcpy(newAirports, thrust::raw_pointer_cast(d_flightZData.data()), d_flightZData.size() * sizeof(int), hipMemcpyDeviceToHost);

		// Copy flights from device to host
		hipError_t errorB = hipMemcpy(hostFlights, d_flights, numFlights * sizeof(Flight), hipMemcpyDeviceToHost);
		if (errorA != hipSuccess || errorB != hipSuccess) {
			std::cerr << "Failed to copy flights to host for updating: " << hipGetErrorString(errorA) << " and/or " << hipGetErrorString(errorB) << std::endl;
			delete[] hostFlights;
			delete[] newAirports;
			return false;
		}

		// Reconstruct the flights on the host
		for (int i = 0; i < numFlights; i++) {
			if (hostFlights[i].position.zLength <= 0) {
				continue;
			}
			hostFlights[i].position.z = newAirports + hostFlights[i].position.zOffset;
		}

		// Do the actual updating of the flights
		for (int i = 0; i < updateCount; i++) {
			int updateIdx = indices[i];
			hostFlights[updateIdx].isRecalculating = false;
			hostFlights[updateIdx].position = newPositions[i];
			hostFlights[updateIdx].flightDuration = newDurations[i];
		}

		// Reinstailize the system
		bool result = initialize(hostFlights, numFlights);

		delete[] hostFlights;
		delete[] newAirports;
		return result;
	}
	else
	{
		// Everything went fine
		// Re-sort flights by X coordinate after update
		indicesDirty = true;
	}

	return true;
}

// Sort flights by X coordinate for efficient sweep
void FlightSystem::sortFlightsByX() {
	// Initialize indices
	thrust::sequence(thrust::device, d_indices, d_indices + numFlights, 0);

	// Sort flights by their x-coordinate
	thrust::sort(thrust::device, d_indices, d_indices + numFlights,
		CompareByX(d_flights));

	findLongestFlightDuration();
	indicesDirty = false;
}

void FlightSystem::findLongestFlightDuration() {
	std::vector<Flight> flight(1);
	Flight* d_maxFlight = thrust::max_element(thrust::device, d_flights, d_flights + numFlights, CompareByDuration());
	hipMemcpy(flight.data(), d_maxFlight, sizeof(Flight), hipMemcpyDeviceToHost);

	longestFlightDuration = flight[0].flightDuration;
}

int* FlightSystem::getMinMaxIndex(int min, int max) {
	// When we sort by time (X) we also need to consider the flight duration.
	// The nicest way to do that is simply to add the longest flight duration to the min value.
	int adjustedMin = min - longestFlightDuration;

	// This requires d_indicies to be sorted by x-coordinate.
	int* lower = thrust::lower_bound(thrust::device, d_indices, d_indices + numFlights, adjustedMin,
		CompareToLowerX(d_flights));
	// Using Lower_bouund with max + 1 here, because upper_bound would not work. Probably a skill issue, but this works. 
	int* higher = thrust::lower_bound(thrust::device, d_indices, d_indices + numFlights, max + 1,
		CompareToLowerX(d_flights));

	// Calculate the indices by subtracting the adresses we get back from lower_bound.
	int lowerIdx = lower - d_indices;
	int upperIdx = higher - d_indices - 1; // -1 because upper_bound gives position after the last element

	int* result = new int[2];
	result[0] = lowerIdx;
	result[1] = upperIdx;

	return result;
}

// Detect collisions with a bounding box
int* FlightSystem::detectCollisions(const BoundingBox& box, bool autoSetRecalculating) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return nullptr;
	}

	if (indicesDirty) {
#if _DEBUG
		std::cout << COLOR_YELLOW << "[DEBUG] Flights not sorted by X, sorting during collision detection" << COLOR_RESET << std::endl;
#endif 
		sortFlightsByX();
	}

	// Binary search to find the first flight that might intersect the box
	int* minMaxIndex = getMinMaxIndex(box.min.x, box.max.x);

	int numFlightsInsideBox = minMaxIndex[1] - minMaxIndex[0] + 1;
	int offset = minMaxIndex[0];

	delete[] minMaxIndex; // Free the memory

	// Uncomment this to scan all flights
	/*int offset = 0;
	int numFlightsInsideBox = numFlights;*/
#if _DEBUG
	std::cout << COLOR_GRAY << "[DEBUG] Saving: " << numFlights - numFlightsInsideBox << " flight lookups through Sort and Sweep" << COLOR_RESET << std::endl;
#endif

	if (numFlightsInsideBox <= 0) {
		return new int[1] { 0 }; // No flights to check, we know they're all outside.
	}

	// Launch collision detection kernel
	int blockSize = 256;
	int numBlocks = (numFlightsInsideBox + blockSize - 1) / blockSize;

	int* zValues = thrust::raw_pointer_cast(d_flightZData.data());
	checkCollisionsKernel << <numBlocks, blockSize >> > (
		d_flights, numFlights, d_indices, zValues, box, offset, autoSetRecalculating, d_collisionResults);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Check for errors
	auto error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "Error detecting collisions: " << hipGetErrorString(error) << std::endl;
		return nullptr;
	}

	// Copy results back to host
	std::vector<int> hostCollisionResults(numFlightsInsideBox);
	error = hipMemcpy(hostCollisionResults.data(), d_collisionResults + offset, numFlightsInsideBox * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy collision results to host: "
			<< hipGetErrorString(error) << std::endl;
		return nullptr;
	}

	// Copy results to output array, skipping the first entry (which will become the length of the array)
	int* collisionResults = new int[numFlightsInsideBox + 1];
	int collisionCount = 0;
	for (int i = 0; i < numFlightsInsideBox; i++) {
		if (hostCollisionResults[i] != INT_MIN) {
			collisionResults[++collisionCount] = hostCollisionResults[i];
		}
	}
	collisionResults[0] = collisionCount;

	return collisionResults;
}

bool FlightSystem::releaseCollisionResults(int* results)
{
	if (results != nullptr) {
		delete[] results;
		return true;
	}
	return false;
}

int FlightSystem::getIndexFromId(int flightId) const {
	auto it = flightIdToIndex.find(flightId);
	return (it != flightIdToIndex.end()) ? it->second : -1;
}

bool FlightSystem::getIndicesFromIds(int* ids, int count, int* indices)
{
	// Would love to make this a kernel function, but CUDA does not have a unordered_map equivalent, so this is the fastest we're going to be.
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	for (int i = 0; i < count; i++) {
		int idx = getIndexFromId(ids[i]);
		if (idx == -1) {
			std::cerr << "Requested flight ID " << ids[i] << " not found" << std::endl;
			return false;
		}
		indices[i] = idx;
	}

	return true;
}

// CUDA kernel to update flight id map
__global__ void fetchFlightIdKernel(Flight* flights, int* idsInOrder, int flightCount) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < flightCount) {
		idsInOrder[idx] = flights[idx].id;
	}
}

// Update the ID to index mapping
void FlightSystem::updateIdToIndexMap() {
	if (numFlights <= 0) {
		flightIdToIndex.clear();
		flightIdMapDirty = false;
		return;
	}

	// Fetch flight data from device to update map
	int* d_idsInOrder;
	hipError_t error = hipMalloc(&d_idsInOrder, numFlights * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for update id to index map: "
			<< hipGetErrorString(error) << std::endl;
		return;
	}

	// Launch collision detection kernel
	int blockSize = 256;
	int numBlocks = (numFlights + blockSize - 1) / blockSize;

	fetchFlightIdKernel << <numBlocks, blockSize >> > (
		d_flights, d_idsInOrder, numFlights);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Check for errors
	error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "Error rebuilding id to index map: " << hipGetErrorString(error) << std::endl;
		return;
	}

	std::vector<int> hostIds(numFlights);
	hipMemcpy(hostIds.data(), d_idsInOrder, numFlights * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_idsInOrder);

	// Clear and rebuild the map
	flightIdToIndex.clear();
	flightIdToIndex.reserve(numFlights);
	for (int i = 0; i < numFlights; i++) {
		flightIdToIndex[hostIds[i]] = i;
	}
	flightIdMapDirty = false;
}

// Free all allocated device memory
void FlightSystem::cleanup() {
	if (d_flights) {
		hipFree(d_flights);
		d_flights = nullptr;
	}

	if (d_indices) {
		hipFree(d_indices);
		d_indices = nullptr;
	}

	if (d_collisionResults) {
		hipFree(d_collisionResults);
		d_collisionResults = nullptr;
	}

	d_flightZData.clear();
	d_flightZData.shrink_to_fit();

	// Free the map memory
	flightIdToIndex.clear();

	initialized = false;
	numFlights = 0;
	allocatedFlights = 0;
}
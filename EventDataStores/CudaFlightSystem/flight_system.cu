#include "hip/hip_runtime.h"
#include "flight_system.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <thrust/extrema.h>
#include <iostream>

// CUDA kernel to update specific flights
__global__ void updateFlightsKernel(Flight* flights, int* indices, FlightPosition* newPositions, int* newDurations, int updateCount) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < updateCount) {
		int flightIdx = indices[idx];
		flights[flightIdx].position = newPositions[idx];
		flights[flightIdx].flightDuration = newDurations[idx];
	}
}

// CUDA kernel to check collisions between flights and a bounding box
__global__ void checkCollisionsKernel(Flight* flights, int numFlights,
	int* indices, BoundingBox box, int offset, int* collisionResults) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < numFlights) {
		int flightIdx = indices[idx + offset];
		FlightPosition dep = flights[flightIdx].position;
		int duration = flights[flightIdx].flightDuration;

		collisionResults[flightIdx] = 0; // Default to no collision

		// We must check if the box intersects with the flight.
		// The flight is a LINE in 3D space made up of points (dep, dest) where dest.x = dep.x + duration
		// Y and Z coordinates are the same. So if they're not in the box, the flight doesn't intersect the box.

		bool yCollision = (dep.y >= box.min.y) && (dep.y <= box.max.y);

		if (!yCollision) {
			return;
		}

		bool zCollision = false;
		for (int i = 0; i < dep.zLength; i++) {
			if (dep.z[i] >= box.min.z && dep.z[i] <= box.max.z) {
				zCollision = true;
				break;
			}
		}

		if (!zCollision) {
			return;
		}

		float xDest = dep.x + duration;
		bool collision =
			((dep.x >= box.min.x) && (dep.x <= box.max.x)) // Checks if flight starts inside the box
			|| (dep.x < box.min.x && xDest > box.min.x); // Checks if flight intersects the box

		collisionResults[flightIdx] = collision ? 1 : 0;
	}
}

// Custom comparison functor for sorting by x position
struct CompareByX {
	Flight* flights;

	CompareByX(Flight* _flights) : flights(_flights) {}

	__host__ __device__ bool operator()(int a, int b) const {
		return flights[a].position.x < flights[b].position.x;
	}
};

// Custom compare function for lower/upper bound search to compare x-coordinate
struct CompareToLowerX {
	Flight* flights;

	CompareToLowerX(Flight* _flights) : flights(_flights) {}

	__host__ __device__ bool operator()(int idx, int val) const {
		return flights[idx].position.x < val;
	}
};

struct CompareByDuration {
	__host__ __device__ bool operator()(Flight a, Flight b)
	{
		return a.flightDuration < b.flightDuration;
	}
};


// Constructor - initialize member variables
FlightSystem::FlightSystem()
	: d_flights(nullptr), d_indices(nullptr), d_collisionResults(nullptr),
	numFlights(0), allocatedFlights(0), initialized(false), deviceId(0) {
	// Get the current CUDA device
	hipGetDevice(&deviceId);
}

// Destructor - cleanup CUDA resources
FlightSystem::~FlightSystem() {
	cleanup();
}

// Allocate or reallocate device memory
bool FlightSystem::allocateDeviceMemory(int requiredSize) {
	// If we already have enough space, no need to reallocate
	if (allocatedFlights >= requiredSize && d_flights != nullptr &&
		d_indices != nullptr && d_collisionResults != nullptr) {
		return true;
	}

	// Calculate new allocation size (with some extra space for future additions)
	int newSize = requiredSize * 1.5; // Allocate 50% extra space
	if (newSize < 100) newSize = 100; // Minimum allocation

	// Allocate new memory
	Flight* new_d_flights = nullptr;
	int* new_d_indices = nullptr;
	int* new_d_collisionResults = nullptr;

	hipError_t error = hipMalloc(&new_d_flights, newSize * sizeof(Flight));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for flights: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	error = hipMalloc(&new_d_indices, newSize * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for indices: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(new_d_flights);
		return false;
	}

	error = hipMalloc(&new_d_collisionResults, newSize * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for collision results: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(new_d_flights);
		hipFree(new_d_indices);
		return false;
	}

	// If we're reallocating, copy existing data to new memory
	if (d_flights != nullptr && numFlights > 0) {
		hipMemcpy(new_d_flights, d_flights, numFlights * sizeof(Flight), hipMemcpyDeviceToDevice);
		hipMemcpy(new_d_indices, d_indices, numFlights * sizeof(int), hipMemcpyDeviceToDevice);

		// Free old memory
		hipFree(d_flights);
		hipFree(d_indices);
		hipFree(d_collisionResults);
	}

	// Set the new pointers
	d_flights = new_d_flights;
	d_indices = new_d_indices;
	d_collisionResults = new_d_collisionResults;
	allocatedFlights = newSize;

	return true;
}

int* FlightSystem::copyToDeviceManaged(int* hostData, int count) {
	int* d_managed;
	hipMalloc(&d_managed, count * sizeof(int));
	managedMallocs.push_back(d_managed);
	auto error = hipMemcpy(d_managed, hostData, count * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		std::cerr << "Failed to managed data to device: "
			<< hipGetErrorString(error) << std::endl;
		return nullptr;
	}
	return d_managed;
}

// Makes a Flight* ready to be copied into device memory. 
void FlightSystem::copyZDataToDeviceManaged(Flight* flights, int count) {
	// Copy all the airports to device
	std::vector<int> airports(count * 3);
	for (int i = 0; i < count; i++) {
		for (int j = 0; j < flights[i].position.zLength; j++) {
			airports.push_back(flights[i].position.z[j]);
		}
	}
	int* d_airports = copyToDeviceManaged(airports.data(), airports.size());
	airports.clear();
	int counter = 0;
	for (int i = 0; i < count; i++) {
		if (flights[i].position.zLength <= 0) {
			flights[i].position.z = nullptr;
			continue;
		}
		flights[i].position.z = d_airports + counter;
		counter += flights[i].position.zLength;
	}
}

// Initialize with flights from host
bool FlightSystem::initialize(Flight* hostFlights, int count) {
	// Clean up previous allocation if any
	cleanup();

	if (count < 0) {
		std::cerr << "Invalid flight count provided for initialization" << std::endl;
		return false;
	}

	numFlights = count;

	// Allocate device memory
	if (!allocateDeviceMemory(count)) {
		return false;
	}

	if (hostFlights != nullptr) {

		// This is a bit flawed because we are making a copy of all incomming flights.
		// I don't nessecarily have to do that, as I could either do it one at the time (saving memory-footprint)
		// or somehow agree with the caller that I will take ownership of the data so I can free any allocated memory that I'm overriding
		std::vector<Flight> flights(hostFlights, hostFlights + count);
		copyZDataToDeviceManaged(flights.data(), count);

		// Copy flights to device
		hipError_t error = hipMemcpy(d_flights, flights.data(), numFlights * sizeof(Flight), hipMemcpyHostToDevice);
		flights.clear();
		if (error != hipSuccess) {
			std::cerr << "Failed to copy flights to device: "
				<< hipGetErrorString(error) << std::endl;
			cleanup();
			return false;
		}

		// Initialize indices and sort flights
		sortFlightsByX();
	}

	initialized = true;
	return true;
}

// Add new flights to the system
bool FlightSystem::addFlights(Flight* newFlights, int count) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (count < 0 || newFlights == nullptr) {
		std::cerr << "Invalid flight data provided for addition" << std::endl;
		return false;
	}

	// Check if we need to reallocate memory
	int newTotal = numFlights + count;
	if (newTotal > allocatedFlights) {
		if (!allocateDeviceMemory(newTotal)) {
			return false;
		}
	}

	// Copy all the airports to device
	std::vector<Flight> flights(newFlights, newFlights + count);
	copyZDataToDeviceManaged(flights.data(), count);

	// Copy new flights to the end of existing flights
	hipError_t error = hipMemcpy(d_flights + numFlights, flights.data(), count * sizeof(Flight), hipMemcpyHostToDevice);
	flights.clear();
	if (error != hipSuccess) {
		std::cerr << "Failed to copy new flights to device: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	// Update flight count
	numFlights = newTotal;

	// Re-sort flights by X coordinate
	sortFlightsByX();

	return true;
}

// Remove flights by indices
bool FlightSystem::removeFlights(int* indices, int count) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (count <= 0 || indices == nullptr) {
		std::cerr << "Invalid indices provided for removal" << std::endl;
		return false;
	}

	// Create a temporary host array of all flights
	Flight* hostFlights = new Flight[numFlights];
	if (!hostFlights) {
		std::cerr << "Failed to allocate host memory for flight removal" << std::endl;
		return false;
	}

	// Copy flights from device to host
	hipError_t error = hipMemcpy(hostFlights, d_flights, numFlights * sizeof(Flight), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy flights to host for removal: " << hipGetErrorString(error) << std::endl;
		delete[] hostFlights;
		return false;
	}

	// Create a flagged array to mark flights for removal
	bool* toRemove = new bool[numFlights]();
	for (int i = 0; i < count; i++) {
		if (indices[i] >= 0 && indices[i] < numFlights) {
			toRemove[indices[i]] = true;
		}
		else {
			std::cerr << "Invalid flight index for removal: " << indices[i] << std::endl;
		}
	}

	// Create a new array without removed flights
	int newCount = 0;
	Flight* newFlights = new Flight[numFlights];

	for (int i = 0; i < numFlights; i++) {
		if (!toRemove[i]) {
			newFlights[newCount++] = hostFlights[i];
			std::vector<int> airports(newFlights[newCount - 1].position.zLength);
			hipMemcpy(airports.data(), newFlights[newCount - 1].position.z, newFlights[newCount - 1].position.zLength * sizeof(int), hipMemcpyDeviceToHost);
			newFlights[newCount - 1].position.z = airports.data();
		}
	}

	// Clean up temporary arrays
	delete[] hostFlights;
	delete[] toRemove;

	// Reinitialize with the new array
	bool result = initialize(newFlights, newCount);

	// Clean up the new array
	delete[] newFlights;

	return result;
}

// Update specific flights with new positions
bool FlightSystem::updateFlights(int* indices, FlightPosition* newPositions, int* newDurations, int updateCount) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (updateCount <= 0 || indices == nullptr || newPositions == nullptr || newDurations == nullptr) {
		std::cerr << "Invalid data provided for flight update" << std::endl;
		return false;
	}

	// Allocate device memory for indices, new positions and durations
	int* d_updateIndices;
	FlightPosition* d_newPositions;
	int* d_newDurations;

	hipError_t error = hipMalloc(&d_updateIndices, updateCount * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for update indices: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	error = hipMalloc(&d_newPositions, updateCount * sizeof(FlightPosition));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for new positions: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		return false;
	}

	error = hipMalloc(&d_newDurations, updateCount * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for new durations: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		hipFree(d_newPositions);
		return false;
	}

	// Copy the airport data over

	std::vector<FlightPosition> positions(newPositions, newPositions + updateCount);
	std::vector<int> airports;
	for (int i = 0; i < updateCount; i++) {
		for (int j = 0; j < positions[i].zLength; j++) {
			airports.push_back(positions[i].z[j]);
		}
	}
	int* d_airports = copyToDeviceManaged(airports.data(), airports.size());
	airports.clear();
	int counter = 0;
	for (int i = 0; i < updateCount; i++) {
		if (positions[i].zLength <= 0) {
			positions[i].z = nullptr;
			continue;
		}
		positions[i].z = d_airports + counter;
		counter += positions[i].zLength;
	}

	// Copy indices and new positions to device
	hipMemcpy(d_updateIndices, indices, updateCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_newPositions, positions.data(), updateCount * sizeof(FlightPosition), hipMemcpyHostToDevice);
	hipMemcpy(d_newDurations, newDurations, updateCount * sizeof(int), hipMemcpyHostToDevice);

	positions.clear();

	// Launch kernel to update flights
	int blockSize = 256;
	int numBlocks = (updateCount + blockSize - 1) / blockSize;

	updateFlightsKernel << <numBlocks, blockSize >> > (
		d_flights, d_updateIndices, d_newPositions, d_newDurations, updateCount);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Check for errors
	error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "Error updating flights: " << hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		hipFree(d_newPositions);
		hipFree(d_newDurations);
		return false;
	}

	// Free temporary device memory
	hipFree(d_updateIndices);
	hipFree(d_newPositions);
	hipFree(d_newDurations);

	// Re-sort flights by X coordinate after update
	sortFlightsByX();

	return true;
}

// Sort flights by X coordinate for efficient sweep
void FlightSystem::sortFlightsByX() {
	// Initialize indices
	thrust::sequence(thrust::device, d_indices, d_indices + numFlights, 0);

	// Sort flights by their x-coordinate
	thrust::sort(thrust::device, d_indices, d_indices + numFlights,
		CompareByX(d_flights));

	findLongestFlightDuration();
}

void FlightSystem::findLongestFlightDuration() {
	std::vector<Flight> flight(1);
	Flight* d_maxFlight = thrust::max_element(thrust::device, d_flights, d_flights + numFlights, CompareByDuration());
	hipMemcpy(flight.data(), d_maxFlight, sizeof(Flight), hipMemcpyDeviceToHost);

	longestFlightDuration = flight[0].flightDuration;
}

int* FlightSystem::getMinMaxIndex(int min, int max) {
	// When we sort by time (X) we also need to consider the flight duration.
	// The nicest way to do that is simply to add the longest flight duration to the min value.
	int adjustedMin = min - longestFlightDuration;

	// This requires d_indicies to be sorted by x-coordinate.
	int* lower = thrust::lower_bound(thrust::device, d_indices, d_indices + numFlights, adjustedMin,
		CompareToLowerX(d_flights));
	// Using Lower_bouund with max + 1 here, because upper_bound would not work. Probably a skill issue, but this works. 
	int* higher = thrust::lower_bound(thrust::device, d_indices, d_indices + numFlights, max + 1,
		CompareToLowerX(d_flights));

	// Calculate the indices by subtracting the adresses we get back from lower_bound.
	int lowerIdx = lower - d_indices;
	int upperIdx = higher - d_indices - 1; // -1 because upper_bound gives position after the last element

	int* result = new int[2];
	result[0] = lowerIdx;
	result[1] = upperIdx;

	return result;
}

// Detect collisions with a bounding box
bool FlightSystem::detectCollisions(const BoundingBox& box, int* collisionResults) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	// Binary search to find the first flight that might intersect the box
	int* minMaxIndex = getMinMaxIndex(box.min.x, box.max.x);

	int numFlightsInsideBox = minMaxIndex[1] - minMaxIndex[0] + 1;
	int offset = minMaxIndex[0];

	delete[] minMaxIndex; // Free the memory

	// Uncomment this to scan all flights
	//offset = 0;
	//numFlightsInsideBox = numFlights;

#if DEBUG
	std::cout << "[DEBUG] Saving: " << numFlights - numFlightsInsideBox << " flight lookups through Sort and Sweep" << std::endl;
#endif 

	if (numFlightsInsideBox <= 0) {
		return true; // No flights to check, we know they're all outside.
	}

	// Launch collision detection kernel
	int blockSize = 256;
	int numBlocks = (numFlightsInsideBox + blockSize - 1) / blockSize;

	checkCollisionsKernel << <numBlocks, blockSize >> > (
		d_flights, numFlights, d_indices, box, offset, d_collisionResults);

	// Wait for kernel to finish
	hipDeviceSynchronize();


	// Check for errors
	auto error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "Error detecting collisions: " << hipGetErrorString(error) << std::endl;
		return false;
	}

	// Copy results back to host
	error = hipMemcpy(collisionResults, d_collisionResults, numFlights * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy collision results to host: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	return true;
}

// Free all allocated device memory
void FlightSystem::cleanup() {
	if (d_flights) {
		hipFree(d_flights);
		d_flights = nullptr;
	}

	if (d_indices) {
		hipFree(d_indices);
		d_indices = nullptr;
	}

	if (d_collisionResults) {
		hipFree(d_collisionResults);
		d_collisionResults = nullptr;
	}

	if (managedMallocs.size() > 0) {
		for (int i = 0; i < managedMallocs.size(); i++) {
			if (managedMallocs[i] != nullptr) {
				hipFree(managedMallocs[i]);
			}
		}
		managedMallocs.clear();
	}

	initialized = false;
	numFlights = 0;
	allocatedFlights = 0;
}
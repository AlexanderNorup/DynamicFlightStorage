#include "hip/hip_runtime.h"
#include "flight_system.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <thrust/extrema.h>
#include <iostream>

// CUDA kernel to update specific flights
__global__ void updateFlightsKernel(Flight* flights, int* indices, Vec3* newPositions, int* newDurations, int updateCount) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < updateCount) {
		int flightIdx = indices[idx];
		flights[flightIdx].position = newPositions[idx];
		flights[flightIdx].flightDuration = newDurations[idx];
	}
}

// CUDA kernel to check collisions between flights and a bounding box
__global__ void checkCollisionsKernel(Flight* flights, int numFlights,
	int* indices, BoundingBox box, int offset, int* collisionResults) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < numFlights) {
		int flightIdx = indices[idx + offset];
		Vec3 dep = flights[flightIdx].position;
		int duration = flights[flightIdx].flightDuration;

		// We must check if the box intersects with the flight.
		// The flight is a LINE in 3D space made up of points (dep, dest) where dest.x = dep.x + duration
		// Y and Z coordinates are the same. So if they're not in the box, the flight doesn't intersect the box.

		bool yzCollision = (dep.y >= box.min.y) && (dep.y <= box.max.y) &&
			(dep.z >= box.min.z) && (dep.z <= box.max.z);

		if (!yzCollision) {
			collisionResults[flightIdx] = 0;
			return;
		}

		float xDest = dep.x + duration;
		bool collision =
			((dep.x >= box.min.x) && (dep.x <= box.max.x)) // Checks if flight starts inside the box
			|| (dep.x < box.min.x && xDest > box.min.x); // Checks if flight intersects the box

		collisionResults[flightIdx] = collision ? 1 : 0;
	}
}

// Custom comparison functor for sorting by x position
struct CompareByX {
	Flight* flights;

	CompareByX(Flight* _flights) : flights(_flights) {}

	__host__ __device__ bool operator()(int a, int b) const {
		return flights[a].position.x < flights[b].position.x;
	}
};

// Custom compare function for lower/upper bound search to compare x-coordinate
struct CompareToLowerX {
	Flight* flights;

	CompareToLowerX(Flight* _flights) : flights(_flights) {}

	__host__ __device__ bool operator()(int idx, int val) const {
		return flights[idx].position.x < val;
	}
};

struct CompareByDuration {
	__host__ __device__ bool operator()(Flight a, Flight b)
	{
		return a.flightDuration < b.flightDuration;
	}
};


// Constructor - initialize member variables
FlightSystem::FlightSystem()
	: d_flights(nullptr), d_indices(nullptr), d_collisionResults(nullptr),
	numFlights(0), allocatedFlights(0), initialized(false), deviceId(0) {
	// Get the current CUDA device
	hipGetDevice(&deviceId);
}

// Destructor - cleanup CUDA resources
FlightSystem::~FlightSystem() {
	cleanup();
}

// Allocate or reallocate device memory
bool FlightSystem::allocateDeviceMemory(int requiredSize) {
	// If we already have enough space, no need to reallocate
	if (allocatedFlights >= requiredSize && d_flights != nullptr &&
		d_indices != nullptr && d_collisionResults != nullptr) {
		return true;
	}

	// Calculate new allocation size (with some extra space for future additions)
	int newSize = requiredSize * 1.5; // Allocate 50% extra space
	if (newSize < 100) newSize = 100; // Minimum allocation

	// Allocate new memory
	Flight* new_d_flights = nullptr;
	int* new_d_indices = nullptr;
	int* new_d_collisionResults = nullptr;

	hipError_t error = hipMalloc(&new_d_flights, newSize * sizeof(Flight));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for flights: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	error = hipMalloc(&new_d_indices, newSize * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for indices: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(new_d_flights);
		return false;
	}

	error = hipMalloc(&new_d_collisionResults, newSize * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for collision results: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(new_d_flights);
		hipFree(new_d_indices);
		return false;
	}

	// If we're reallocating, copy existing data to new memory
	if (d_flights != nullptr && numFlights > 0) {
		hipMemcpy(new_d_flights, d_flights, numFlights * sizeof(Flight), hipMemcpyDeviceToDevice);
		hipMemcpy(new_d_indices, d_indices, numFlights * sizeof(int), hipMemcpyDeviceToDevice);

		// Free old memory
		hipFree(d_flights);
		hipFree(d_indices);
		hipFree(d_collisionResults);
	}

	// Set the new pointers
	d_flights = new_d_flights;
	d_indices = new_d_indices;
	d_collisionResults = new_d_collisionResults;
	allocatedFlights = newSize;

	return true;
}

// Initialize with flights from host
bool FlightSystem::initialize(Flight* hostFlights, int count) {
	// Clean up previous allocation if any
	cleanup();

	if (count < 0) {
		std::cerr << "Invalid flight count provided for initialization" << std::endl;
		return false;
	}

	numFlights = count;

	// Allocate device memory
	if (!allocateDeviceMemory(count)) {
		return false;
	}

	if (hostFlights != nullptr) {
		// Copy flights to device
		hipError_t error = hipMemcpy(d_flights, hostFlights, numFlights * sizeof(Flight), hipMemcpyHostToDevice);
		if (error != hipSuccess) {
			std::cerr << "Failed to copy flights to device: "
				<< hipGetErrorString(error) << std::endl;
			cleanup();
			return false;
		}

		// Initialize indices and sort flights
		sortFlightsByX();
	}

	initialized = true;
	return true;
}

// Add new flights to the system
bool FlightSystem::addFlights(Flight* newFlights, int count) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (count < 0 || newFlights == nullptr) {
		std::cerr << "Invalid flight data provided for addition" << std::endl;
		return false;
	}

	// Check if we need to reallocate memory
	int newTotal = numFlights + count;
	if (newTotal > allocatedFlights) {
		if (!allocateDeviceMemory(newTotal)) {
			return false;
		}
	}

	// Copy new flights to the end of existing flights
	hipError_t error = hipMemcpy(d_flights + numFlights, newFlights, count * sizeof(Flight), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy new flights to device: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	// Update flight count
	numFlights = newTotal;

	// Re-sort flights by X coordinate
	sortFlightsByX();

	return true;
}

// Remove flights by indices
bool FlightSystem::removeFlights(int* indices, int count) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (count <= 0 || indices == nullptr) {
		std::cerr << "Invalid indices provided for removal" << std::endl;
		return false;
	}

	// Create a temporary host array of all flights
	Flight* hostFlights = new Flight[numFlights];
	if (!hostFlights) {
		std::cerr << "Failed to allocate host memory for flight removal" << std::endl;
		return false;
	}

	// Copy flights from device to host
	hipError_t error = hipMemcpy(hostFlights, d_flights, numFlights * sizeof(Flight), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy flights to host for removal: " << hipGetErrorString(error) << std::endl;
		delete[] hostFlights;
		return false;
	}

	// Create a flagged array to mark flights for removal
	bool* toRemove = new bool[numFlights]();
	for (int i = 0; i < count; i++) {
		if (indices[i] >= 0 && indices[i] < numFlights) {
			toRemove[indices[i]] = true;
		}
		else {
			std::cerr << "Invalid flight index for removal: " << indices[i] << std::endl;
		}
	}

	// Create a new array without removed flights
	int newCount = 0;
	Flight* newFlights = new Flight[numFlights];

	for (int i = 0; i < numFlights; i++) {
		if (!toRemove[i]) {
			newFlights[newCount++] = hostFlights[i];
		}
	}

	// Clean up temporary arrays
	delete[] hostFlights;
	delete[] toRemove;

	// Reinitialize with the new array
	bool result = initialize(newFlights, newCount);

	// Clean up the new array
	delete[] newFlights;

	return result;
}

// Update specific flights with new positions
bool FlightSystem::updateFlights(int* indices, Vec3* newPositions, int* newDurations, int updateCount) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	if (updateCount <= 0 || indices == nullptr || newPositions == nullptr || newDurations == nullptr) {
		std::cerr << "Invalid data provided for flight update" << std::endl;
		return false;
	}

	// Allocate device memory for indices, new positions and durations
	int* d_updateIndices;
	Vec3* d_newPositions;
	int* d_newDurations;

	hipError_t error = hipMalloc(&d_updateIndices, updateCount * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for update indices: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	error = hipMalloc(&d_newPositions, updateCount * sizeof(Vec3));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for new positions: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		return false;
	}

	error = hipMalloc(&d_newDurations, updateCount * sizeof(int));
	if (error != hipSuccess) {
		std::cerr << "Failed to allocate device memory for new durations: "
			<< hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		hipFree(d_newPositions);
		return false;
	}

	// Copy indices and new positions to device
	hipMemcpy(d_updateIndices, indices, updateCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_newPositions, newPositions, updateCount * sizeof(Vec3), hipMemcpyHostToDevice);
	hipMemcpy(d_newDurations, newDurations, updateCount * sizeof(int), hipMemcpyHostToDevice);

	// Launch kernel to update flights
	int blockSize = 256;
	int numBlocks = (updateCount + blockSize - 1) / blockSize;

	updateFlightsKernel << <numBlocks, blockSize >> > (
		d_flights, d_updateIndices, d_newPositions, d_newDurations, updateCount);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Check for errors
	error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "Error updating flights: " << hipGetErrorString(error) << std::endl;
		hipFree(d_updateIndices);
		hipFree(d_newPositions);
		hipFree(d_newDurations);
		return false;
	}

	// Free temporary device memory
	hipFree(d_updateIndices);
	hipFree(d_newPositions);
	hipFree(d_newDurations);

	// Re-sort flights by X coordinate after update
	sortFlightsByX();

	return true;
}

// Sort flights by X coordinate for efficient sweep
void FlightSystem::sortFlightsByX() {
	// Initialize indices
	thrust::sequence(thrust::device, d_indices, d_indices + numFlights, 0);

	// Sort flights by their x-coordinate
	thrust::sort(thrust::device, d_indices, d_indices + numFlights,
		CompareByX(d_flights));

	findLongestFlightDuration();
}

void FlightSystem::findLongestFlightDuration() {
	std::vector<Flight> flight(1);
	Flight* d_maxFlight = thrust::max_element(thrust::device, d_flights, d_flights + numFlights, CompareByDuration());
	hipMemcpy(flight.data(), d_maxFlight, sizeof(Flight), hipMemcpyDeviceToHost);

	longestFlightDuration = flight[0].flightDuration;
}

int* FlightSystem::getMinMaxIndex(int min, int max) {
	// When we sort by time (X) we also need to consider the flight duration.
	// The nicest way to do that is simply to add the longest flight duration to the min value.
	int adjustedMin = min - longestFlightDuration;

	// This requires d_indicies to be sorted by x-coordinate.
	int* lower = thrust::lower_bound(thrust::device, d_indices, d_indices + numFlights, adjustedMin,
		CompareToLowerX(d_flights));
	// Using Lower_bouund with max + 1 here, because upper_bound would not work. Probably a skill issue, but this works. 
	int* higher = thrust::lower_bound(thrust::device, d_indices, d_indices + numFlights, max + 1,
		CompareToLowerX(d_flights));

	// Calculate the indices by subtracting the adresses we get back from lower_bound.
	int lowerIdx = lower - d_indices;
	int upperIdx = higher - d_indices - 1; // -1 because upper_bound gives position after the last element

	int* result = new int[2];
	result[0] = lowerIdx;
	result[1] = upperIdx;

	return result;
}

// Detect collisions with a bounding box
bool FlightSystem::detectCollisions(const BoundingBox& box, int* collisionResults) {
	if (!initialized) {
		std::cerr << "Flight system not initialized" << std::endl;
		return false;
	}

	// Binary search to find the first flight that might intersect the box
	int* minMaxIndex = getMinMaxIndex(box.min.x, box.max.x);

	int numFlightsInsideBox = minMaxIndex[1] - minMaxIndex[0] + 1;
	int offset = minMaxIndex[0];

	delete[] minMaxIndex; // Free the memory

	// Uncomment this to scan all flights
	//offset = 0;
	//numFlightsInsideBox = numFlights;

#if DEBUG
	std::cout << "[DEBUG] Saving: " << numFlights - numFlightsInsideBox << " flight lookups through Sort and Sweep" << std::endl;
#endif 

	if (numFlightsInsideBox <= 0) {
		return true; // No flights to check, we know they're all outside.
	}

	// Launch collision detection kernel
	int blockSize = 256;
	int numBlocks = (numFlightsInsideBox + blockSize - 1) / blockSize;

	checkCollisionsKernel << <numBlocks, blockSize >> > (
		d_flights, numFlights, d_indices, box, offset, d_collisionResults);

	// Wait for kernel to finish
	hipDeviceSynchronize();


	// Check for errors
	auto error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "Error detecting collisions: " << hipGetErrorString(error) << std::endl;
		return false;
	}

	// Copy results back to host
	error = hipMemcpy(collisionResults, d_collisionResults, numFlights * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cerr << "Failed to copy collision results to host: "
			<< hipGetErrorString(error) << std::endl;
		return false;
	}

	return true;
}

// Free all allocated device memory
void FlightSystem::cleanup() {
	if (d_flights) {
		hipFree(d_flights);
		d_flights = nullptr;
	}

	if (d_indices) {
		hipFree(d_indices);
		d_indices = nullptr;
	}

	if (d_collisionResults) {
		hipFree(d_collisionResults);
		d_collisionResults = nullptr;
	}

	initialized = false;
	numFlights = 0;
	allocatedFlights = 0;
}